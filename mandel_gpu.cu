#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <R.h>

#define ITERATIONS 200
#define BLOCK_X 16
#define BLOCK_Y 16

typedef struct {
    double re;
    double im;
} complex;

__device__ complex zero()
{
    complex res;
    res.re = 0.0;
    res.im = 0.0;
    return res;
}

__device__ complex add(complex a, complex b) 
{
    complex res;
    res.re = a.re + b.re;
    res.im = a.im + b.im;
    return res;
}

__device__ complex mul(complex a, complex b)
{
    complex res;
    res.re = a.re * b.re - a.im * b.im;
    res.im = a.re * b.im + a.im * b.re;
    return res;
}

__device__ double square_mod(complex c)
{
    return c.re * c.re + c.im * c.im;
}

__device__ double color(complex c)
{
    complex z = zero();
    for (int i = 1; i <= ITERATIONS; ++i) {
        if (square_mod(z) >= 4.0) {
            return (1.0 - (double)i / ITERATIONS);
        }
        z = add(mul(z, z), c);
    }
    return 0.0;
}

__global__ void kernel(double *buffer, int cols, int rows, double sr, double dr, double si, double di)
{
    int col = threadIdx.x + blockIdx.x * blockDim.x;
    int row = threadIdx.y + blockIdx.y * blockDim.y;

    int offset = row + col * rows;

    if (offset < cols * rows) {

        complex c;
        c.re = sr + dr * col;
        c.im = si + di * row;

        buffer[offset] = color(c);
    }
}

extern "C" void mandel_gpu(int *w, int *h, double *sr, double *er, double *si, double *ei, double *result) 
{
    hipError_t err;
    double* buffer = 0;
    err = hipMalloc((void**)&buffer, (*w) * (*h) * sizeof(double));
    if (err != hipSuccess) {
        Rprintf("hipMalloc error: %d [%s]\n", err, hipGetErrorString(err));
        return;
    }

    double dr = (*er - *sr) / *w;
    double di = (*ei - *si) / *h;

    dim3 blocks( ((*w) + BLOCK_X - 1) / BLOCK_X, ((*h) + BLOCK_Y - 1) / BLOCK_Y);
    dim3 threads(BLOCK_X, BLOCK_Y);
    kernel<<<blocks, threads>>>(buffer, *w, *h, *sr, dr, *si, di);

    err = hipMemcpy(result, buffer, (*w) * (*h) * sizeof(double), hipMemcpyDeviceToHost);
    hipFree(buffer);
    if (err != hipSuccess) {
        Rprintf("hipMemcpy error: %d [%s]\n", err, hipGetErrorString(err));
    }
}